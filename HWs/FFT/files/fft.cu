#include "hip/hip_runtime.h"
//ONLY MODIFY THIS FILE!
//YOU CAN MODIFY EVERYTHING IN THIS FILE!

#include "fft.h"

#define tx threadIdx.x
#define ty threadIdx.y
#define tz threadIdx.z

#define bx blockIdx.x
#define by blockIdx.y
#define bz blockIdx.z

#define Radix (4)
#define gridDim (1<<10)
#define blockDim (1<<6)
#define BETA (blockDim*1)
#define ALPHA (gridDim*1)
#define uint unsigned int

#define THREAD_PER_BLOCK_SORT 1024 
#define BLOCK_X  1024
#define THREAD_PER_BLOCK_FLY  256  

// you may define other parameters here!
// you may define other macros here!
// you may define other functions here!
// __global__ void sharedMemoryFft(float* x_r_d, float* x_i_d, const unsigned int N, const unsigned int M)
// {
// 	__shared__ float x_shared_r[BETA];
// 	__shared__ float x_shared_i[BETA];//could more than one element
	

	

// }

__global__ void gpuSort(float* x, float* x_temp, const unsigned int M)
{
	uint index = tx + ( by * BLOCK_X + bx  ) * THREAD_PER_BLOCK_SORT;
	uint reversed2bit = index;
	reversed2bit = (reversed2bit & 0xFFFF0000) >> 16 | (reversed2bit & 0x0000FFFF) << 16;
	reversed2bit = (reversed2bit & 0xFF00FF00) >>  8 | (reversed2bit & 0x00FF00FF) <<  8;
	reversed2bit = (reversed2bit & 0xF0F0F0F0) >>  4 | (reversed2bit & 0x0F0F0F0F) <<  4;
	reversed2bit = (reversed2bit & 0xAAAAAAAA) >>  2 | (reversed2bit & 0x33333333) <<  2;
	reversed2bit = reversed2bit >> (32-M);
	x_temp[index] = x[reversed2bit];
}

__global__ void gpuCopy(float* x, float* x_temp)
{
	uint index = tx + ( by * BLOCK_X + bx  ) * THREAD_PER_BLOCK_SORT;
	x[index] = x_temp[index];
}

__global__ void gpuButterfly(float* x_r_d, float* x_i_d, uint v)
{
	uint index = tx + ( by * BLOCK_X + bx  ) * THREAD_PER_BLOCK_FLY / 2;
	uint p = index % (1<<v);
	uint fl = 1<<v;
	uint indexNew = (index/fl)*(4*fl) + p;
	float const theta = (PI * p)/(2*fl);
	float cosinus, sinus;
	float x_i_temp, x_r_temp;


	float w0_real, w0_imag, w1_real, w1_imag, w2_real, w2_imag, w3_real, w3_imag;
	w0_real = x_r_d[indexNew];
	w0_imag = x_i_d[indexNew];
	
	indexNew += fl;
	x_r_temp = x_r_d[indexNew];
	x_i_temp = x_i_d[indexNew];
	sincosf(theta, &sinus, &cosinus);
	w1_real = cosinus * x_r_temp + sinus * x_i_temp;
	w1_imag = cosinus * x_i_temp - sinus * x_r_temp;

	indexNew += fl;
	float theta_temp = theta + theta;
	x_r_temp = x_r_d[indexNew];
	x_i_temp = x_i_d[indexNew];
	sincosf(theta_temp, &sinus, &cosinus);
	w2_real = cosinus * x_r_temp + sinus * x_i_temp;
	w2_imag = cosinus * x_i_temp - sinus * x_r_temp;

	indexNew += fl;
	theta_temp += theta;
	x_r_temp = x_r_d[indexNew];
	x_i_temp = x_i_d[indexNew];
	sincosf(theta_temp, &sinus, &cosinus);
	w3_real = cosinus * x_r_temp + sinus * x_i_temp;
	w3_imag = cosinus * x_i_temp - sinus * x_r_temp;

	x_r_d[index] = w0_real - w1_imag - w2_real + w3_imag;
	x_i_d[index] = w0_imag + w1_real - w2_imag - w3_real;
	index += fl;
	x_r_d[index] = w0_real - w1_real + w2_real - w3_real;
	x_i_d[index] = w0_imag - w1_imag + w2_imag - w3_imag;
	index += fl;
	x_r_d[index] = w0_real + w1_imag - w2_real - w3_imag;
	x_i_d[index] = w0_imag - w1_real - w2_imag + w3_real;
	index += fl;
	x_r_d[index] = w0_real + w1_real + w2_real + w3_real;
	x_i_d[index] = w0_imag + w1_imag + w2_imag + w3_imag;
}

//-----------------------------------------------------------------------------
void gpuKernel(float* x_r_d, float* x_i_d, const unsigned int N, const unsigned int M)
{
	float *d_temp;
	HANDLE_ERROR(hipMalloc((void**) & d_temp, N * sizeof(float)));
	dim3 dimGrid_Swap(BLOCK_X,N/BLOCK_X/THREAD_PER_BLOCK_SORT); 
	dim3 dimBlock_Swap(THREAD_PER_BLOCK_SORT,1);
	dim3 dimGrid_Butterfly(BLOCK_X,N/4/BLOCK_X/(THREAD_PER_BLOCK_FLY/2)); 
	dim3 dimBlock_Butterfly(THREAD_PER_BLOCK_FLY/2,1);
	gpuSort<<< dimGrid_Swap,dimBlock_Swap >>>(x_r_d, d_temp, M);
	gpuCopy<<< dimGrid_Swap,dimBlock_Swap >>>(x_r_d, d_temp);
	gpuSort<<< dimGrid_Swap,dimBlock_Swap >>>(x_i_d, d_temp, M);
	gpuCopy<<< dimGrid_Swap,dimBlock_Swap >>>(x_i_d, d_temp);
	HANDLE_ERROR(hipFree(d_temp));
	for(uint i = 0; i < M; i += 2)
		gpuButterfly<<< dimGrid_Butterfly ,dimBlock_Butterfly >>>(x_r_d,x_i_d,i);


	// In this function, both inputs and outputs are on GPU.
	// No need for hipMalloc, hipMemcpy or hipFree.
	// This function does not run on GPU. 
	// You need to define another function and call it here for GPU execution.
	
}
