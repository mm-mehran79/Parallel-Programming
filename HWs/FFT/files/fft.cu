//ONLY MODIFY THIS FILE!
//YOU CAN MODIFY EVERYTHING IN THIS FILE!

#include "fft.h"

#define tx threadIdx.x
#define ty threadIdx.y
#define tz threadIdx.z

#define bx blockIdx.x
#define by blockIdx.y
#define bz blockIdx.z

// you may define other parameters here!
// you may define other macros here!
// you may define other functions here!

//-----------------------------------------------------------------------------
void gpuKernel(float* x_r_d, float* x_i_d, /*float* X_r_d, float* X_i_d,*/ const unsigned int N, const unsigned int M)
{
	// In this function, both inputs and outputs are on GPU.
	// No need for hipMalloc, hipMemcpy or hipFree.
	
}
