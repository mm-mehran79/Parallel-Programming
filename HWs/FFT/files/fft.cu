#include "hip/hip_runtime.h"
//ONLY MODIFY THIS FILE!
//YOU CAN MODIFY EVERYTHING IN THIS FILE!

#include "fft.h"

#define tx threadIdx.x
#define ty threadIdx.y
#define tz threadIdx.z

#define bx blockIdx.x
#define by blockIdx.y
#define bz blockIdx.z

#define Radix (4)
#define gridDim (1<<10)
#define blockDim (1<<6)
#define BETA (blockDim*1)
#define ALPHA (gridDim*1)
#define uint unsigned int

#define THREAD_PER_BLOCK_SORT 1024 
#define BLOCK_X  1024
#define THREAD_PER_BLOCK_FLY  256  

//Replace all sincosf with __sincosf for faster speed but lower accuracy (and vice versa)

// you may define other parameters here!
// you may define other macros here!
// you may define other functions here!
// __global__ void sharedMemoryFft(float* x_r_d, float* x_i_d, const unsigned int N, const unsigned int M)
// {
// 	__shared__ float x_shared_r[BETA];
// 	__shared__ float x_shared_i[BETA];//could more than one element
	

	

// }

__global__ void gpuSort(float* x, float* x_temp, const unsigned int N, const unsigned int M)
{
	// int id = tx + ( by * BLOCK_X + bx  ) * THREAD_PER_BLOCK_SORT;
	// int new_id = id;
	// int half = N/4;
	// int b = 0;

	// for(int i = 0 ; i<M; i++ ){
	// 	if( new_id % 4 == 0 ){
	// 		new_id = (new_id-b)/4 + b;
    // 	}
    // 	else if ( new_id % 4 == 1 ) {
	// 		new_id = (new_id-b)/4 + half + b;
	// 		b += half ;
    // 	}
	// 	else if ( new_id % 4 == 2 ) {
	// 		new_id = (new_id-b)/4 + 2*half + b;
	// 		b += 2*half ;
    // 	}
	// 	else{
	// 		new_id = (new_id-b)/4 + 3*half + b;
	// 		b += 3*half ;
    // 	}
	// 	half /= 4;
	// }
	// x_temp[new_id]=x[id];
	///////////////////// 2nd way:
	// int index = tx + ( by * BLOCK_X + bx  ) * THREAD_PER_BLOCK_SORT;
	// int reversed2bit = 0, index_copy = index;
	// for(int i = 0; i < M; i++)
	// {
	// 	reversed2bit *= 4;
	// 	reversed2bit += index_copy%4;
	// 	index_copy /= 4;
	// }
	// uint reversed2bit = index;
	// uint reversed2bit = (index & 0xFFFF0000U) >> 16 | (index & 0x0000FFFFU) << 16;
	// uint reversed2bit1 = (reversed2bit & 0xFF00FF00U) >>  8 | (reversed2bit & 0x00FF00FFU) <<  8;
	// uint reversed2bit2 = (reversed2bit1 & 0xF0F0F0F0U) >>  4 | (reversed2bit1 & 0x0F0F0F0FU) <<  4;
	// uint reversed2bit3 = (reversed2bit2 & 0xCCCCCCCCU) >>  2 | (reversed2bit2 & 0x33333333U) <<  2;
	// uint reversed2bit4 = reversed2bit3 >> (32-M);
	// x_temp[index] = x[reversed2bit4];
	//////////////////////////////// 4th way:
	uint index = tx + ( by * BLOCK_X + bx  ) * THREAD_PER_BLOCK_SORT;
	uint indexBitReversed = __brev(index);
	uint index2BitReversed = (indexBitReversed & 0xAAAAAAAAU) >>  1 | (indexBitReversed & 0x55555555U) <<  1;
	uint index2BitReversedOut = index2BitReversed >> (32U-M);
	x_temp[index2BitReversedOut] = x[index];
}

__global__ void gpuSortRADIX2(float* x, float* x_temp, const unsigned int N, const unsigned int M)
{
	uint index = tx + ( by * BLOCK_X + bx  ) * THREAD_PER_BLOCK_SORT;
	uint indexBitReversed = __brev(index);
	uint indexBitReversedOut = indexBitReversed >> (32U-M);
	x_temp[indexBitReversedOut] = x[index];
}

__global__ void gpuCopy(float* x, float* x_temp)
{
	uint index = tx + ( by * BLOCK_X + bx  ) * THREAD_PER_BLOCK_SORT;
	x[index] = x_temp[index];
}

__global__ void gpuButterfly(float* x_r_d, float* x_i_d, uint v)
{
	uint index = tx + ( by * BLOCK_X + bx  ) * THREAD_PER_BLOCK_FLY / 2;
	uint fl = 1<<v;
	uint p = index % fl;
	uint indexNew = (index/fl)*(4*fl) + p;
	index = indexNew;
	float const theta = (PI * p)/(2*fl);
	float cosinus, sinus;
	float x_i_temp, x_r_temp;


	float w0_real, w0_imag, w1_real, w1_imag, w2_real, w2_imag, w3_real, w3_imag;
	w0_real = x_r_d[indexNew];
	w0_imag = x_i_d[indexNew];
	
	indexNew += fl;
	x_r_temp = x_r_d[indexNew];
	x_i_temp = x_i_d[indexNew];
	__sincosf(theta, &sinus, &cosinus);
	w1_real = cosinus * x_r_temp + sinus * x_i_temp;
	w1_imag = cosinus * x_i_temp - sinus * x_r_temp;

	indexNew += fl;
	float theta_temp = theta + theta;
	x_r_temp = x_r_d[indexNew];
	x_i_temp = x_i_d[indexNew];
	__sincosf(theta_temp, &sinus, &cosinus);
	w2_real = cosinus * x_r_temp + sinus * x_i_temp;
	w2_imag = cosinus * x_i_temp - sinus * x_r_temp;

	indexNew += fl;
	theta_temp += theta;
	x_r_temp = x_r_d[indexNew];
	x_i_temp = x_i_d[indexNew];
	__sincosf(theta_temp, &sinus, &cosinus);
	w3_real = cosinus * x_r_temp + sinus * x_i_temp;
	w3_imag = cosinus * x_i_temp - sinus * x_r_temp;

	x_r_d[index] = w0_real + w1_real + w2_real + w3_real;
	x_i_d[index] = w0_imag + w1_imag + w2_imag + w3_imag;
	index += fl;
	x_r_d[index] = w0_real + w1_imag - w2_real - w3_imag;
	x_i_d[index] = w0_imag - w1_real - w2_imag + w3_real;
	index += fl;
	x_r_d[index] = w0_real - w1_real + w2_real - w3_real;
	x_i_d[index] = w0_imag - w1_imag + w2_imag - w3_imag;
	index += fl;
	x_r_d[index] = w0_real - w1_imag - w2_real + w3_imag;
	x_i_d[index] = w0_imag + w1_real - w2_imag - w3_real;
}

__global__ void gpuButterflyRADIX2(float* x_r_d, float* x_i_d, uint v)
{
	uint index = tx + ( by * BLOCK_X + bx  ) * THREAD_PER_BLOCK_FLY;
	uint fl = 1<<v;
	uint p = index % fl;
	uint indexNew = (index/fl)*(2*fl) + p;
	index = indexNew;
	float const theta = (PI * p)/(fl);
	float cosinus, sinus;
	float x_i_temp, x_r_temp;


	float w0_real, w0_imag, w1_real, w1_imag;
	w0_real = x_r_d[indexNew];
	w0_imag = x_i_d[indexNew];
	
	indexNew += fl;
	x_r_temp = x_r_d[indexNew];
	x_i_temp = x_i_d[indexNew];
	__sincosf(theta, &sinus, &cosinus);
	w1_real = cosinus * x_r_temp + sinus * x_i_temp;
	w1_imag = cosinus * x_i_temp - sinus * x_r_temp;

	x_r_d[index] = w0_real + w1_real;
	x_i_d[index] = w0_imag + w1_imag;
	index += fl;
	x_r_d[index] = w0_real - w1_real;
	x_i_d[index] = w0_imag - w1_imag;
}

//-----------------------------------------------------------------------------
void gpuKernel(float* x_r_d, float* x_i_d, const unsigned int N, const unsigned int M)
{
	float *d_temp;
	HANDLE_ERROR(hipMalloc((void**) & d_temp, N * sizeof(float)));
	if (M%2)
	{
		//radix 2
		dim3 dimGrid_Swap(BLOCK_X,N/BLOCK_X/THREAD_PER_BLOCK_SORT); 
		dim3 dimBlock_Swap(THREAD_PER_BLOCK_SORT,1);
		gpuSortRADIX2<<< dimGrid_Swap,dimBlock_Swap >>>(x_r_d, d_temp, N, M);
		gpuCopy<<< dimGrid_Swap,dimBlock_Swap >>>(x_r_d, d_temp);

		dim3 dimGrid_Butterfly(BLOCK_X,N/2/BLOCK_X/(THREAD_PER_BLOCK_FLY)); 
		dim3 dimBlock_Butterfly(THREAD_PER_BLOCK_FLY,1);
		gpuSortRADIX2<<< dimGrid_Swap,dimBlock_Swap >>>(x_i_d, d_temp, N, M);
		gpuCopy<<< dimGrid_Swap,dimBlock_Swap >>>(x_i_d, d_temp);

		for(uint i = 0; i < M; i += 1)
			gpuButterflyRADIX2<<< dimGrid_Butterfly ,dimBlock_Butterfly >>>(x_r_d,x_i_d,i);
		HANDLE_ERROR(hipFree(d_temp));
	}
	else
	{
		//radix 4
		dim3 dimGrid_Swap(BLOCK_X,N/BLOCK_X/THREAD_PER_BLOCK_SORT); 
		dim3 dimBlock_Swap(THREAD_PER_BLOCK_SORT,1);
		gpuSort<<< dimGrid_Swap,dimBlock_Swap >>>(x_r_d, d_temp, N, M);
		gpuCopy<<< dimGrid_Swap,dimBlock_Swap >>>(x_r_d, d_temp);
		
		dim3 dimGrid_Butterfly(BLOCK_X,N/4/BLOCK_X/(THREAD_PER_BLOCK_FLY/2)); 
		dim3 dimBlock_Butterfly(THREAD_PER_BLOCK_FLY/2,1);
		gpuSort<<< dimGrid_Swap,dimBlock_Swap >>>(x_i_d, d_temp, N, M);
		gpuCopy<<< dimGrid_Swap,dimBlock_Swap >>>(x_i_d, d_temp);
		
		
		for(uint i = 0; i < M; i += 2)
			gpuButterfly<<< dimGrid_Butterfly ,dimBlock_Butterfly >>>(x_r_d,x_i_d,i);
		HANDLE_ERROR(hipFree(d_temp));
	}
	

		
	// In this function, both inputs and outputs are on GPU.
	// No need for hipMalloc, hipMemcpy or hipFree.
	// This function does not run on GPU. 
	// You need to define another function and call it here for GPU execution.
	
}
