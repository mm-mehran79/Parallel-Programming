#include "hip/hip_runtime.h"
//ONLY MODIFY THIS FILE!
//YOU CAN MODIFY EVERYTHING IN THIS FILE!

#include "fft.h"

#define tx threadIdx.x
#define ty threadIdx.y
#define tz threadIdx.z

#define bx blockIdx.x
#define by blockIdx.y
#define bz blockIdx.z

#define Radix (4)
#define gridDim (1<<10)
#define blockDim (1<<6)
#define BETA (blockDim*1)
#define ALPHA (gridDim*1)
#define uint unsigned int

#define THREAD_PER_BLOCK_SORT 1024 
#define BLOCK_X  1024
#define THREAD_PER_BLOCK_FLY  256  

// you may define other parameters here!
// you may define other macros here!
// you may define other functions here!
// __global__ void sharedMemoryFft(float* x_r_d, float* x_i_d, const unsigned int N, const unsigned int M)
// {
// 	__shared__ float x_shared_r[BETA];
// 	__shared__ float x_shared_i[BETA];//could more than one element
	

	

// }

__global__ void gpuSort(float* x, float* x_temp, const unsigned int N, const unsigned int M)
{
	// int id = tx + ( by * BLOCK_X + bx  ) * THREAD_PER_BLOCK_SORT;
	// int new_id = id;
	// int half = N/4;
	// int b = 0;

	// for(int i = 0 ; i<M; i++ ){
	// 	if( new_id % 4 == 0 ){
	// 		new_id = (new_id-b)/4 + b;
    // 	}
    // 	else if ( new_id % 4 == 1 ) {
	// 		new_id = (new_id-b)/4 + half + b;
	// 		b += half ;
    // 	}
	// 	else if ( new_id % 4 == 2 ) {
	// 		new_id = (new_id-b)/4 + 2*half + b;
	// 		b += 2*half ;
    // 	}
	// 	else{
	// 		new_id = (new_id-b)/4 + 3*half + b;
	// 		b += 3*half ;
    // 	}
	// 	half /= 4;
	// }
	// x_temp[new_id]=x[id];
	///////////////////// 2nd way:
	// int index = tx + ( by * BLOCK_X + bx  ) * THREAD_PER_BLOCK_SORT;
	// int reversed2bit = 0, index_copy = index;
	// for(int i = 0; i < M; i++)
	// {
	// 	reversed2bit *= 4;
	// 	reversed2bit += index_copy%4;
	// 	index_copy /= 4;
	// }
	// uint reversed2bit = index;
	// uint reversed2bit = (index & 0xFFFF0000U) >> 16 | (index & 0x0000FFFFU) << 16;
	// uint reversed2bit1 = (reversed2bit & 0xFF00FF00U) >>  8 | (reversed2bit & 0x00FF00FFU) <<  8;
	// uint reversed2bit2 = (reversed2bit1 & 0xF0F0F0F0U) >>  4 | (reversed2bit1 & 0x0F0F0F0FU) <<  4;
	// uint reversed2bit3 = (reversed2bit2 & 0xCCCCCCCCU) >>  2 | (reversed2bit2 & 0x33333333U) <<  2;
	// uint reversed2bit4 = reversed2bit3 >> (32-M);
	// x_temp[index] = x[reversed2bit4];
	//////////////////////////////// 4th way:
	uint index = tx + ( by * BLOCK_X + bx  ) * THREAD_PER_BLOCK_SORT;
	uint indexBitReversed = __brev(index);
	uint index2BitReversed = (indexBitReversed & 0xAAAAAAAAU) >>  1 | (indexBitReversed & 0x55555555U) <<  1;
	uint index2BitReversedOut = index2BitReversed >> (32U-M);
	x_temp[index2BitReversedOut] = x[index];
}

__global__ void gpuCopy(float* x, float* x_temp)
{
	uint index = tx + ( by * BLOCK_X + bx  ) * THREAD_PER_BLOCK_SORT;
	x[index] = x_temp[index];
}

__global__ void gpuButterfly(float* x_r_d, float* x_i_d, uint v)
{
	// uint fl = 1<<v;
	// int id = tx + ( by * BLOCK_X + bx  ) * THREAD_PER_BLOCK_FLY / 2;
    // unsigned int sh = id  % fl;
	// id = (id/fl) * (4*fl)  + sh;

	// float wr = x_r_d[id];
	// float wi = x_i_d[id];
	// float arg;

	// id=id+fl;
	// float x_r_1 = x_r_d[id];
	// float x_i_1 = x_i_d[id];
	// arg = (PI * sh )/(2*fl);
	// float cos_r, sin_r;
    // // float cos_r = cos(arg);
	// // float sin_r = sin(arg);
	// __sincosf(arg, &sin_r, &cos_r);
	// float wr_1 = cos_r*x_r_1+sin_r*x_i_1;
	// float wi_1 = -sin_r*x_r_1+cos_r*x_i_1;

	// arg=arg*2;
    // // cos_r = cos(arg);
    // // sin_r = sin(arg);
	// __sincosf(arg, &sin_r, &cos_r);
	// id=id+fl;
	// x_r_1 = x_r_d[id];
	// x_i_1 = x_i_d[id];
	// float wr_2 = cos_r*x_r_1+sin_r*x_i_1;
	// float wi_2 = -sin_r*x_r_1+cos_r*x_i_1;

	// arg=arg*3.0/2.0;
    // // cos_r = cos(arg);
    // // sin_r = sin(arg);
	// __sincosf(arg, &sin_r, &cos_r);
	// id=id+fl;
	// x_r_1 = x_r_d[id];
	// x_i_1 = x_i_d[id];
	// float wr_3 = cos_r*x_r_1+sin_r*x_i_1;
	// float wi_3 = -sin_r*x_r_1+cos_r*x_i_1;

	// x_r_d[id] = wr - wi_1 - wr_2 + wi_3;
	// x_i_d[id] = wi + wr_1 - wi_2 - wr_3;
	// id=id-fl;
	// x_r_d[id] = wr - wr_1 + wr_2 - wr_3;
	// x_i_d[id] = wi - wi_1 + wi_2 - wi_3;
	// id=id-fl;
	// x_r_d[id]     = wr + wi_1 - wr_2 - wi_3;
	// x_i_d[id]     = wi - wr_1 - wi_2 + wr_3;
	// id=id-fl;
	// x_r_d[id]          = wr + wr_1 + wr_2 + wr_3;
	// x_i_d[id]          = wi + wi_1 + wi_2 + wi_3;
	///////////////////////////////////////////////////// 2nd way:

	uint index = tx + ( by * BLOCK_X + bx  ) * THREAD_PER_BLOCK_FLY / 2;
	uint fl = 1<<v;
	uint p = index % fl;
	uint indexNew = (index/fl)*(4*fl) + p;
	index = indexNew;
	float const theta = (PI * p)/(2*fl);
	float cosinus, sinus;
	float x_i_temp, x_r_temp;


	float w0_real, w0_imag, w1_real, w1_imag, w2_real, w2_imag, w3_real, w3_imag;
	w0_real = x_r_d[indexNew];
	w0_imag = x_i_d[indexNew];
	
	indexNew += fl;
	x_r_temp = x_r_d[indexNew];
	x_i_temp = x_i_d[indexNew];
	sincosf(theta, &sinus, &cosinus);
	w1_real = cosinus * x_r_temp + sinus * x_i_temp;
	w1_imag = cosinus * x_i_temp - sinus * x_r_temp;

	indexNew += fl;
	float theta_temp = theta + theta;
	x_r_temp = x_r_d[indexNew];
	x_i_temp = x_i_d[indexNew];
	sincosf(theta_temp, &sinus, &cosinus);
	w2_real = cosinus * x_r_temp + sinus * x_i_temp;
	w2_imag = cosinus * x_i_temp - sinus * x_r_temp;

	indexNew += fl;
	theta_temp += theta;
	x_r_temp = x_r_d[indexNew];
	x_i_temp = x_i_d[indexNew];
	sincosf(theta_temp, &sinus, &cosinus);
	w3_real = cosinus * x_r_temp + sinus * x_i_temp;
	w3_imag = cosinus * x_i_temp - sinus * x_r_temp;

	x_r_d[index] = w0_real + w1_real + w2_real + w3_real;
	x_i_d[index] = w0_imag + w1_imag + w2_imag + w3_imag;
	index += fl;
	x_r_d[index] = w0_real + w1_imag - w2_real - w3_imag;
	x_i_d[index] = w0_imag - w1_real - w2_imag + w3_real;
	index += fl;
	x_r_d[index] = w0_real - w1_real + w2_real - w3_real;
	x_i_d[index] = w0_imag - w1_imag + w2_imag - w3_imag;
	index += fl;
	x_r_d[index] = w0_real - w1_imag - w2_real + w3_imag;
	x_i_d[index] = w0_imag + w1_real - w2_imag - w3_real;
}

//-----------------------------------------------------------------------------
void gpuKernel(float* x_r_d, float* x_i_d, const unsigned int N, const unsigned int M)
{
	float *d_temp;
	HANDLE_ERROR(hipMalloc((void**) & d_temp, N * sizeof(float)));

	dim3 dimGrid_Swap(BLOCK_X,N/BLOCK_X/THREAD_PER_BLOCK_SORT); 
	dim3 dimBlock_Swap(THREAD_PER_BLOCK_SORT,1);
	gpuSort<<< dimGrid_Swap,dimBlock_Swap >>>(x_r_d, d_temp, N, M);
	gpuCopy<<< dimGrid_Swap,dimBlock_Swap >>>(x_r_d, d_temp);
	
	dim3 dimGrid_Butterfly(BLOCK_X,N/4/BLOCK_X/(THREAD_PER_BLOCK_FLY/2)); 
	dim3 dimBlock_Butterfly(THREAD_PER_BLOCK_FLY/2,1);
	gpuSort<<< dimGrid_Swap,dimBlock_Swap >>>(x_i_d, d_temp, N, M);
	gpuCopy<<< dimGrid_Swap,dimBlock_Swap >>>(x_i_d, d_temp);
	
	// HANDLE_ERROR(hipFree(d_temp));
	for(uint i = 0; i < M; i += 2)
		gpuButterfly<<< dimGrid_Butterfly ,dimBlock_Butterfly >>>(x_r_d,x_i_d,i);
	HANDLE_ERROR(hipFree(d_temp));
	// In this function, both inputs and outputs are on GPU.
	// No need for hipMalloc, hipMemcpy or hipFree.
	// This function does not run on GPU. 
	// You need to define another function and call it here for GPU execution.
	
}
