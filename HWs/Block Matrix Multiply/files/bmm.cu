#include "hip/hip_runtime.h"
// ONLY MODIFY THIS FILE!
// YOU CAN MODIFY EVERYTHING IN THIS FILE!

#include "bmm.h"

#define tx threadIdx.x
#define ty threadIdx.y
#define tz threadIdx.z

#define bx blockIdx.x
#define by blockIdx.y
#define bz blockIdx.z

// TILEX and TILEY are used to set the number of threads in a CUDA block
#define TILEX 32
#define TILEY 16

// DEPH is used to set size of array in shared memmory
#define DEPTH 128

// you may define other parameters here!
// you may define other macros here!
//#define MOD(x, y) ((x) & ((1 << (y)) - 1))
// #define MOD(x, y) ((x) % ((1 << (y))))
// you may define other functions here!

dim3 getDimGrid(const int m, const int n)
{
	dim3 dimGrid(n / TILEX, n / TILEY);
	return dimGrid;
}
dim3 getDimBlock(const int m, const int n)
{
	dim3 dimBlock(TILEX, TILEY);
	return dimBlock;
}
__global__ void kernelFunc(float *ad, float *bd, float *cd, const int m, const int n)
{
	__shared__ float As[TILEY][DEPTH];
	__shared__ float Bs[DEPTH][TILEX];
	float tempC = 0;
	int k, l;
	int tyk, txk;
	const int i = ty + (by * TILEY);
	const int j = tx + (bx * TILEX);
	for (k = 0; k < n; k += DEPTH)
	{
		txk = k + tx;
		for (l = 0; l < DEPTH; l += TILEX)
			As[ty][tx + l] = mem2d(ad, m, i, txk + l);
		tyk = k + ty;
		for (l = 0; l < DEPTH; l += TILEY)
			Bs[ty + l][tx] = mem2d(bd, m, tyk + l, j);
		__syncthreads();
		
		for (l = 0; l < DEPTH; ++l)
		{
			tempC += As[ty][l] * Bs[l][tx];
		}
		__syncthreads();
	}
	mem2d(cd, m, i, j) = tempC;
}
