#include "hip/hip_runtime.h"
// ONLY MODIFY THIS FILE!
// YOU CAN MODIFY EVERYTHING IN THIS FILE!

#include "bmm.h"

#define tx threadIdx.x
#define ty threadIdx.y
#define tz threadIdx.z

#define bx blockIdx.x
#define by blockIdx.y
#define bz blockIdx.z

#define TILEXSHIFT 5
#define TILEYSHIFT 5
#define DEPTHSHIFT 7
// TILEX and TILEY are used to set the number of threads in a CUDA block
#define TILEX (1 << TILEXSHIFT)
#define TILEY (1 << TILEYSHIFT)

// DEPH is used to set size of array in shared memmory
#define DEPTH (1 << DEPTHSHIFT)

// you may define other parameters here!
// you may define other macros here!
//#define MOD(x, y) ((x) & ((1 << (y)) - 1))
// #define MOD(x, y) ((x) % ((1 << (y))))
// you may define other functions here!

dim3 getDimGrid(const int m, const int n)
{
	dim3 dimGrid(n / TILEX, n / TILEY);
	return dimGrid;
}
dim3 getDimBlock(const int m, const int n)
{
	dim3 dimBlock(TILEX, TILEY);
	return dimBlock;
}
__global__ void kernelFunc(float *ad, float *bd, float *cd, const int m, const int n)
{
	__shared__ float As[TILEY][DEPTH];
	__shared__ float Bs[DEPTH][TILEX];
	float tempC = 0;
	unsigned short k, l;
	const unsigned short i = ty + (by << TILEYSHIFT);
	const unsigned short j = tx + (bx << TILEXSHIFT);
	for (k = 0; k < (n >> DEPTHSHIFT); k++)
	{
		__syncthreads();
		#if DEPTH == TILEX
			As[ty][tx] = mem2d(ad, m, i, (k<<DEPTHSHIFT) + tx);
		#elif DEPTH > TILEX
			for (l = 0; l < (1 << (DEPTHSHIFT-TILEXSHIFT)); l++)
				As[ty][tx + (l<<TILEXSHIFT)] = mem2d(ad, m, i, (k << DEPTHSHIFT) + tx + (l<<TILEXSHIFT));
		#else
			if ( (tx >> DEPTHSHIFT) == 0 )
				As[ty][tx] = mem2d(ad, m, i, (k << DEPTHSHIFT) + tx);
		#endif

		#if DEPTH == TILEY
			Bs[ty][tx] = mem2d(bd, m, (k<<DEPTHSHIFT) + ty, j);
		#elif DEPTH > TILEY
			for (l = 0; l < (1 << (DEPTHSHIFT-TILEYSHIFT)); l++)
				Bs[ty + (l<<TILEYSHIFT)][tx] = mem2d(bd, m, (k << DEPTHSHIFT) + ty + (l<<TILEYSHIFT), j);
		#else
			if ( (ty >> DEPTHSHIFT) == 0 )
				Bs[ty][tx] = mem2d(bd, m, (k << DEPTHSHIFT) + ty, j);
		#endif
		__syncthreads();
		for (l = 0; l < DEPTH; l++)
		{
			tempC += As[ty][l] * Bs[l][tx];
		}
	}
	mem2d(cd, m, i, j) = tempC;
}
