#include "hip/hip_runtime.h"
//ONLY MODIFY THIS FILE!
//YOU CAN MODIFY EVERYTHING IN THIS FILE!

#include "bmm.h"

#define tx threadIdx.x
#define ty threadIdx.y
#define tz threadIdx.z

#define bx blockIdx.x
#define by blockIdx.y
#define bz blockIdx.z

// TILEX and TILEY are used to set the number of threads in a CUDA block 
#define TILEX 16
#define TILEY 8

// you may define other parameters here!
// you may define other macros here!
// you may define other functions here!

dim3 getDimGrid(const int m, const int n) {
	dim3 dimGrid(n/TILEX,n/TILEY);
	return dimGrid;
}
dim3 getDimBlock(const int m, const int n) {
	dim3 dimBlock(TILEX,TILEY);
	return dimBlock;
}
__global__ void kernelFunc(float* ad, float* bd, float* cd, const int m, const int n) {

	// write your GPU kernel function here
}
