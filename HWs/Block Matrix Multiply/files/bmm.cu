#include "hip/hip_runtime.h"
// ONLY MODIFY THIS FILE!
// YOU CAN MODIFY EVERYTHING IN THIS FILE!

#include "bmm.h"

#define tx threadIdx.x
#define ty threadIdx.y
#define tz threadIdx.z

#define bx blockIdx.x
#define by blockIdx.y
#define bz blockIdx.z
#define _UI16_MAX 0xffff

#define TILEXSHIFT 4
#define TILEYSHIFT 4
#define DEPTHSHIFT 3
// TILEX and TILEY are used to set the number of threads in a CUDA block
#define TILEX (1 << TILEXSHIFT)
#define TILEY (1 << TILEYSHIFT)

// DEPH is used to set size of array in shared memmory
#define DEPTH (1 << DEPTHSHIFT)

// you may define other parameters here!
// you may define other macros here!
// you may define other functions here!

dim3 getDimGrid(const int m, const int n)
{
	dim3 dimGrid(n / TILEX, n / TILEY);
	return dimGrid;
}
dim3 getDimBlock(const int m, const int n)
{
	dim3 dimBlock(TILEX, TILEY);
	return dimBlock;
}
__global__ void kernelFunc(float *ad, float *bd, float *cd, const int m, const int n)
{
	__shared__ float As[TILEY][DEPTH];
	__shared__ float Bs[DEPTH][TILEX];
	float tempC = 0;
	unsigned short k, l;
	unsigned short i = ty + (by << TILEYSHIFT);
	unsigned short j = tx + (bx << TILEXSHIFT);
	for (k = 0; k < (n >> DEPTHSHIFT); k++)
	{
		__syncthreads();
		if ( (tx >> DEPTHSHIFT) == (k & (~(_UI16_MAX<<(TILEXSHIFT - DEPTHSHIFT)))) )			// tx / DEPTH == (k % TILEX) / DEPTH
			As[ty][tx & (~(_UI16_MAX << DEPTHSHIFT)) ] = mem2d(ad, m, i, k << DEPTHSHIFT + (ty & (~(_UI16_MAX << DEPTHSHIFT))) );
		if ( (ty >> DEPTHSHIFT) == (k & (~(_UI16_MAX<<(TILEYSHIFT - DEPTHSHIFT)))) )
			Bs[ ty & (~(_UI16_MAX << DEPTHSHIFT)) ][tx] = mem2d(bd, m, k << DEPTHSHIFT + (ty & (~(_UI16_MAX << DEPTHSHIFT))), j);
		__syncthreads();

		for (l = 0; l < DEPTH; l++)
		{
			tempC += As[ty][l] * Bs[l][tx];
		}
	}
	mem2d(cd, m, i, j) = tempC;

	// write your GPU kernel function here
}
